#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<iostream>

#include <stdio.h>



__device__ double fun(double x) {
	//return x;
	//return x*x/2+3*x-1/x;
	return (.5 * x * x) + (3 * x) - (1 / x);
}



__global__ void sumintegral(double lowbound1, int n, double dx,double *d_c)
{
	
	//double c = 0;
	for (int i = 0; i < n; i++) {
		double xi = lowbound1 + (i * dx);
		double funValue = fun(xi);
		double rectangleArea = funValue * dx;
		*d_c += rectangleArea;

	}
	//printf("%f ",c);
	
}


int main() {
	
	double lowbound1 = 3;	
	long int n = 1;	
	double c;
	double *d_c;
	double size = sizeof(double);
	hipMalloc((void**)&d_c, size);
	c = 0;
	hipMemcpy(&d_c, &c, size, hipMemcpyHostToDevice);
	
	double dx = (double)lowbound1/n;	
	 sumintegral << <1, 1 >> > (lowbound1, n, dx,d_c);
	 hipMemcpy(&c, &d_c, size, hipMemcpyDeviceToHost);
	 hipFree(&d_c);
	 printf("%f", c);
    return 0;
}