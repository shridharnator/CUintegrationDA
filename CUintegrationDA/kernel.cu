#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<iostream>
//#include<stdlib.h>

#include <stdio.h>



__device__ double fun(double x) {
	//return x;
	//return x*x/2+3*x-1/x;
	return (.5 * x * x) + (3 * x) - (1 / x);
}



__global__ void sumintegral(double lowbound1, int n, double dx,double *d_c)
{
	
	//double c = 0;
	int i = threadIdx.x;
	//if(i<n) {
		/*double xi = lowbound1 + (i * dx);
		double funValue = fun(xi);
		double rectangleArea = funValue * dx;
		*d_c += rectangleArea;*/
		*d_c += 1;

		printf("love");
	//}
	//printf("%f ",c);
	
}


int main() {
	
	double lowbound1 = 3;	
	long int n = 10;	
	double c;

	c = 0;
	double *d_c;
	double size = sizeof(double);
	
	hipMalloc((void**)&d_c, size);
	d_c = &c;
	//std::cout << *d_c << std::endl;
	
	//hipMemcpy(d_c, &c, size, hipMemcpyHostToDevice);
	
	double dx = (double)lowbound1/n;	
	 sumintegral << <1, 10 >> > (lowbound1, n, dx,d_c);
	 hipDeviceSynchronize();
	// hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost); 
	 c = *d_c;
	
	 printf("%f", c);
	 hipFree(&d_c);
    return 0;
}