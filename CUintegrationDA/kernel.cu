#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<iostream>

#include <stdio.h>



__device__ double fun(double x) {
	//return x;
	//return x*x/2+3*x-1/x;
	return (.5 * x * x) + (3 * x) - (1 / x);
}



__global__ void sumintegral(double lowbound1, int n, double dx)
{
	
	double c = 0;
	for (int i = 0; i < n; i++) {
		double xi = lowbound1 + (i * dx);
		double funValue = fun(xi);
		double rectangleArea = funValue * dx;
		c += rectangleArea;

	}
	printf("%f ",c);
	
}


int main() {
	
	double lowbound1 = 3;	
	long int n = 1;	
	double dx = (double)lowbound1/n;	
	 sumintegral << <1, 1 >> > (lowbound1, n, dx);
    return 0;
}